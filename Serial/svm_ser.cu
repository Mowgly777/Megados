//#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helper_functions.h"

// CUDA helper functions
#include "hip/hip_runtime_api.h" 

//------------------DATA SETUP-------------------

const unsigned int numClasses = 10;
const unsigned int T = 10000;
const char *filenameTrain = "train_data_1024.csv";
const char *filenameTest = "train_data_1024.csv";
const char *filenameTrainClassSizes = "class_sizes_train_1024.csv";
const char *filenameTestClassSizes = "class_sizes_train_1024.csv";
const unsigned int c = 1024;
const unsigned int line_len = 35000;

//-----------------------------------------------

//https://stackoverflow.com/questions/9210528/split-string-with-delimiters-in-c
char** str_split(char* a_str, const char a_delim)
{
    char** result    = 0;
    size_t count     = 0;
    char* tmp        = a_str;
    char* last_comma = 0;
    char delim[2];
    delim[0] = a_delim;
    delim[1] = 0;

    /* Count how many elements will be extracted. */
    while (*tmp)
    {
        if (a_delim == *tmp)
        {
            count++;
            last_comma = tmp;
        }
        tmp++;
    }

    /* Add space for trailing token. */
    count += last_comma < (a_str + strlen(a_str) - 1);

    /* Add space for terminating null string so caller
       knows where the list of returned strings ends. */
    count++;

    result = (char**)malloc(sizeof(char*) * count);

    if (result)
    {
        size_t idx  = 0;
        char* token = strtok(a_str, delim);

        while (token)
        {
            assert(idx < count);
            *(result + idx++) = strdup(token);
            token = strtok(0, delim);
        }
        assert(idx == count - 1);
        *(result + idx) = 0;
    }

    return result;
}

/*
X, y enter the method as null pointers.
-The necessary data is extracted and stored in X.
-The y values are also generated according to the classSizes.
*/
void getModelData(float *data, float *X, float *y, int *classSizes, int cl1_id, int cl2_id){
	
	int cl1_start_idx = classSizes[cl1_id*2 + 0];
	int cl1_end_idx = cl1_start_idx + classSizes[cl1_id*2 + 1];
	int cl2_start_idx = classSizes[cl2_id*2 + 0];
	int cl2_end_idx = cl2_start_idx + classSizes[cl2_id*2 + 1];
	int k = 0;
	
	for(int i = cl1_start_idx; i<cl1_end_idx; i++){
		for(int j = 0; j<c; j++){
			X[k*c + j] = data[i*c + j];
		}
		y[k] = 1;
		k++;
	}
	
	for(int i = cl2_start_idx; i<cl2_end_idx; i++){
		for(int j = 0; j<c; j++){
			X[k*c + j] = data[i*c + j];
		}
		y[k] = -1;
		k++;
	}
}

void dotProd(float *xi, float *W, float *result){

	result[0] = 0.0;
	
	for(int k = 0; k<c; k++){
		result[0] += xi[k]*W[k];	
	}
	
}

void updateWeights(float *W, float n, float y, float alpha, float *x, int violatesMC){
	for(int i = 0; i<c; i++){
		W[i] = (1.0 - n*alpha)*W[i] + n*y*x[i]*violatesMC;
	}
}

void pegaFit(float *X, float *y, float *W, unsigned int T, float alpha, unsigned int r,float *runTime,StopWatchInterface *timer){

	float *xi = (float *)malloc(c * sizeof(float)); 
	float yi;
	float *result = (float *)malloc(sizeof(float));
	float ni;
	srand(time(NULL));
	float time= sdkGetTimerValue(&timer);
	for(int i = 1; i <= T; i++){
	
		ni = 1.0/(alpha*(i));
		
		int idx = rand() % r;
		//bringing in current x and y
		for(int k = 0; k<c; k++){
			xi[k] = X[idx*c + k];
		}
		yi = y[idx];
		
		//if violates margin constraint 0.0 and 1.0 are multiplier to nulify second factor
		
		dotProd(xi, W, result);
		
		if(!(yi*(result[0]) >= 1)){
			updateWeights(W, ni, yi, alpha, xi, 1.0);
		}
		else{
			updateWeights(W, ni, yi, alpha, xi, 0.0);
		}
	}
	runTime[0] = sdkGetTimerValue(&timer) - time;
	//printf("weight[0]: %f\n",W[0]);
	
}

void voting(float *xi, float *W, float *voteMat){


	float *dot_calc = (float *)malloc(sizeof(float));
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
		
			dotProd(xi, &W[j*numClasses*c + k*c+0], dot_calc);
			if (dot_calc[0] < 0){
				voteMat[j*numClasses+k+0] = -1;
			} else {
				voteMat[j*numClasses+k+0] = 1;
			}
		}
		
	}
	
	
}

int decision(float *voteMat){

	int vote_count[numClasses] = {0};
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
			if (voteMat[j*numClasses+k] < 0)
				vote_count[k] += 1;
			else
				vote_count[j] += 1;
		}
		
	}
	
	
	
	int max_ind=0;
	for(int i=0; i < numClasses;i++){
		if(vote_count[i] > vote_count[max_ind])
			max_ind = i;
	}
	
	/*for(int loop1 =0; loop1<numClasses;loop1++){
		printf("%d\t",vote_count[loop1]);
	}
	printf("\n");
	*/
	return max_ind;
}

int main(int argc, char **argv){
	
	//-----------------------DATA READ TRAIN------------------------
	
	FILE* stream = fopen(filenameTrainClassSizes, "r");
	
	//class sizes holds the number of rows in each class, 
	//as well as what row number the class begins
	int *classSizes = (int *)malloc(numClasses * 2 * sizeof(int)); 
	
    char line[line_len];
	unsigned int i = 0;
	int numRecords=0;
    while (fgets(line, line_len, stream))
	{
	
		classSizes[i*2 + 0] = numRecords;
		classSizes[i*2 + 1] = atoi(line);
		numRecords += classSizes[i*2+1];
		i++;
    }
	
	stream = fopen(filenameTrain, "r");
	float *data = (float *)malloc(numRecords * c * sizeof(float));
		
	i = 0;
	char** tokens;
	while (fgets(line, line_len, stream))
	{
	    tokens = str_split(line, ',');
	    
	    for(int l = 0; l<c; l++){
			data[i*c + l] = atof(tokens[l]);
			
		}
		i++;
	}
	
	//-----------------------DATA READ TEST------------------------
	stream = fopen(filenameTestClassSizes, "r");
	
	//class sizes holds the number of rows in each class, 
	//as well as what row number the class begins
	int *classSizesTe = (int *)malloc(numClasses * 2 * sizeof(int)); 
	
    
	i = 0;
	int numRecordsTe=0;
    while (fgets(line, line_len, stream))
	{
	
		classSizesTe[i*2 + 0] = numRecordsTe;
		classSizesTe[i*2 + 1] = atoi(line);
		numRecordsTe += classSizesTe[i*2+1];
		i++;
    }
	
	stream = fopen(filenameTest, "r");
	float *dataTe = (float *)malloc(numRecordsTe * c * sizeof(float));
		
	i = 0;
	char** tokensTe;
	while (fgets(line, line_len, stream))
	{
	    tokensTe = str_split(line, ',');
	    
	    for(int l = 0; l<c; l++){
			dataTe[i*c + l] = atof(tokensTe[l]);
		}
		i++;
	}
	//-----------------------WEIGHT MATRIX---------------------
	
	
	float *W = (float *)malloc(numClasses * numClasses * c * sizeof(float));
	float *voteMat = (float *)malloc(numClasses * numClasses * sizeof(float));
	float *time = (float *)malloc(sizeof(float));
	time[0] = 0;
	//-----------------------SERIAL PEGASOS--------------------- 
	
	StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
			
		
			int trainRows = classSizes[j*2 + 1] + classSizes[k*2 + 1];
			float *train_X = (float *)malloc(trainRows * c * sizeof(float));
			float *train_y = (float *)malloc(trainRows * sizeof(float));
			
			getModelData(data, train_X, train_y, classSizes, j, k);
			
			pegaFit(train_X, train_y, &W[j*numClasses*c+k*c+0], T, 0.0001/*alpha*/, trainRows, time, timer);
		}
	}
	
	sdkStopTimer(&timer);
	float tot_time = sdkGetTimerValue(&timer);
    printf("Total time      :\t%f(ms)\n", tot_time);
    printf("Processing time :\t%f(ms)\n", time[0]);
    printf("Overheads time  :\t%f(ms)\n", tot_time-time[0]);
    sdkDeleteTimer(&timer);
	
	int dec=0;
	int numCorr=0;
	for(int j=0; j<numRecordsTe; j++){
		voting(&dataTe[j*c], W, voteMat);
		//printf("rec:%d\t",j);
		dec = decision(voteMat);
		
		for(int k=0;k<numClasses;k++){
			if(j>classSizesTe[k*2+0] && j < (classSizesTe[k*2 + 0]+classSizesTe[k*2 + 1]) ){
			
				if(k == dec){
					numCorr++;
				} else {
					//printf("Dec: %d \tExpected: %d\n",dec,k);
					/*for(int loop1 =0; loop1<numClasses;loop1++){
						for(int loop2=0;loop2<numClasses;loop2++)
							printf("%f\t",voteMat[loop1*numClasses+loop2]);
						printf("\n");
					}*/
							
				}
				
				break;
			}
		}
	}
	float avg = (numCorr/(numRecordsTe+0.0))*100;
	printf("Correct Percentage: %f\n",avg);
}
