#include "hip/hip_runtime.h"
//#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helper_functions.h"

// CUDA helper functions
#include "hip/hip_runtime_api.h" 

//------------------DATA SETUP-------------------

const unsigned int numClasses = 10;
const unsigned int T = 10000;
const char *filenameTrain = "train_data_1024.csv";
const char *filenameTest = "train_data_1024.csv";
const char *classSizeFile ="class_sizes_train_1024.csv";
const unsigned int c = 1024;
const unsigned int line_len = 35000;



//-----------------------------------------------

//https://stackoverflow.com/questions/9210528/split-string-with-delimiters-in-c
char** str_split(char* a_str, const char a_delim)
{
    char** result    = 0;
    size_t count     = 0;
    char* tmp        = a_str;
    char* last_comma = 0;
    char delim[2];
    delim[0] = a_delim;
    delim[1] = 0;

    /* Count how many elements will be extracted. */
    while (*tmp)
    {
        if (a_delim == *tmp)
        {
            count++;
            last_comma = tmp;
        }
        tmp++;
    }

    /* Add space for trailing token. */
    count += last_comma < (a_str + strlen(a_str) - 1);

    /* Add space for terminating null string so caller
       knows where the list of returned strings ends. */
    count++;

    result = (char**)malloc(sizeof(char*) * count);

    if (result)
    {
        size_t idx  = 0;
        char* token = strtok(a_str, delim);

        while (token)
        {
            assert(idx < count);
            *(result + idx++) = strdup(token);
            token = strtok(0, delim);
        }
        assert(idx == count - 1);
        *(result + idx) = 0;
    }

    return result;
}

/*
X, y enter the method as null pointers.
-The necessary data is extracted and stored in X.
-The y values are also generated according to the classSizes.
*/
void getModelData(float *data, float *X, float *y, int *classSizes, int cl1_id, int cl2_id){
	
	int cl1_start_idx = classSizes[cl1_id*2 + 0];
	int cl1_end_idx = cl1_start_idx + classSizes[cl1_id*2 + 1];
	int cl2_start_idx = classSizes[cl2_id*2 + 0];
	int cl2_end_idx = cl2_start_idx + classSizes[cl2_id*2 + 1];
	int k = 0;
	
	for(int i = cl1_start_idx; i<cl1_end_idx; i++){
		for(int j = 0; j<c; j++){
			X[k*c + j] = data[i*c + j];
		}
		y[k] = 1;
		k++;
	}
	
	for(int i = cl2_start_idx; i<cl2_end_idx; i++){
		for(int j = 0; j<c; j++){
			X[k*c + j] = data[i*c + j];
		}
		y[k] = -1;
		k++;
	}
}

void dotProd(float *xi, float *W, float *result){

	//result[0] = 0.0;
	
	for(int k = 0; k<c; k++){
		result[0] += xi[k]*W[k];	
	}
	
}

__global__ void dotprodPar(float *X, float *W, float *ni , float *y, int *pos){
	
	__shared__ float partialSum[c];
	__shared__ int mult;
	
	unsigned int idx = threadIdx.y+threadIdx.x;
	
	//printf("ni[0]:%f\n",ni[0]);
	
	partialSum[idx] = X[(pos[0])*c+idx] * W[idx];
	
	__syncthreads();
	
	for(int stride = blockDim.x/2; stride >= 1; stride = stride >> 1){
		__syncthreads();	
		if(idx < stride){
			partialSum[idx] += partialSum[idx + stride];
		}
	}
	
	__syncthreads();
	
	if(idx == 0){
		if(!(y[pos[0]]*(partialSum[0]) >= 1))
			mult=1;
		else
			mult=0;
	}
	__syncthreads();
	
	W[idx] = (1.0 - ni[0]*0.0001)*W[idx] + ni[0]*y[pos[0]]*X[(pos[0])*c+idx]*mult;
	
	__syncthreads();
	
	if(idx == 0){
		ni[0] = 1.0/(0.0001*pos[1]);
	}
	
		
}


void pegaFit(float *X, float *y, float *W, unsigned int T, float alpha, int r, float *runTime,StopWatchInterface *timer){

	float *result = (float *)malloc(c * sizeof(float));
	float *ni = (float *)malloc(sizeof(float));
	srand(time(NULL));
	
	//parrallel setup
	
    float *x_par = NULL;
    checkCudaErrors(hipMalloc((void **) &x_par, c * r * sizeof(float)));
	float *w_par = NULL;
	checkCudaErrors(hipMalloc((void **) &w_par, c * sizeof(float)));
	float *ni_par = NULL;
    checkCudaErrors(hipMalloc((void **) &ni_par, sizeof(float)));
	float *yi_par = NULL;
	checkCudaErrors(hipMalloc((void **) &yi_par, r * sizeof(float)));
	//int *r_par = NULL;
	//checkCudaErrors(hipMalloc((void **) &r_par, sizeof(int)));
	//end setup
	
	ni[0] = 1.0/(alpha*1);
	
	int *pos = (int *)malloc(2*sizeof(int));
	int *pos_par = NULL;
	hipMalloc((void **) &pos_par, 2*sizeof(int));
	/*
	malloc the full X and Y data
	generate random guess inside dotprodpar
	paralelise voting & decision
	make accuracy a method
	*/
	hipMemcpy(x_par, X, c * r * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yi_par, y, r * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(w_par, W, c * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ni_par, ni, sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(r_par, &r, sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy()
	/*
	StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    */
    float time= sdkGetTimerValue(&timer);
	for(int i = 1; i <= T; i++){
		
		pos[0] = rand()%r;
		pos[1] = i;
		//printf("%d\n",pos[0]);
		hipMemcpy(pos_par, pos, 2*sizeof(int), hipMemcpyHostToDevice);
		dotprodPar<<<1, c>>>(x_par, w_par, ni_par,yi_par,pos_par);
		checkCudaErrors(hipDeviceSynchronize());
	}
	runTime[0] = sdkGetTimerValue(&timer) - time;
	/*
	sdkStopTimer(&timer);
    runTime[0] += sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
	*/
	hipMemcpy(W, w_par, c * sizeof(float), hipMemcpyDeviceToHost);
	
	//printf("weight[0]: %f\n",W[0]);
	
}

void voting(float *xi, float *W, float *voteMat){


	float *dot_calc = (float *)malloc(sizeof(float));
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
			
			dot_calc[0] = 0;
			dotProd(xi, &W[j*numClasses*c + k*c+0], dot_calc);
			if (dot_calc[0] < 0){
				voteMat[j*numClasses+k+0] = -1;
			} else {
				voteMat[j*numClasses+k+0] = 1;
			}
		}
		
	}
	
	
}

int decision(float *voteMat){

	int vote_count[numClasses] = {0};
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
			if (voteMat[j*numClasses+k] < 0)
				vote_count[k] += 1;
			else
				vote_count[j] += 1;
		}
		
	}
	int max_ind=0;
	for(int i=0; i < numClasses;i++){
		if(vote_count[i] > vote_count[max_ind])
			max_ind = i;
	}
	return max_ind;
}

int numCorrect(int numRecords, float *data, float *W, int *classSizes){
	float *voteMat = (float *)malloc(numClasses * numClasses * sizeof(float));
	int numCorr= 0;
	int dec=0;
	
	for(int j=0; j<numRecords; j++){
		voting(&data[j*c], W, voteMat);
		dec = decision(voteMat);
		
		for(int k=0;k<numClasses;k++){
			if(j>classSizes[k*2+0] && j < (classSizes[k*2 + 0]+classSizes[k*2 + 1]) ){
			
				if(k == dec){
					numCorr++;
				}
				break;
			}
		}
	}
	return numCorr;
}

int main(int argc, char **argv){
	
	//-----------------------DATA READ------------------------
	
	FILE* stream = fopen(classSizeFile, "r");
	
	//class sizes holds the number of rows in each class, 
	//as well as what row number the class begins
	int *classSizes = (int *)malloc(numClasses * 2 * sizeof(int)); 
	
    char line[line_len];
	unsigned int i = 0;
	int numRecords=0;
    while (fgets(line, line_len, stream))
	{
	
		classSizes[i*2 + 0] = numRecords;
		classSizes[i*2 + 1] = atoi(line);
		numRecords += classSizes[i*2+1];
		i++;
    }
	
	stream = fopen(filenameTrain, "r");
	float *data = (float *)malloc(numRecords * c * sizeof(float));
		
	i = 0;
	char** tokens;
	while (fgets(line, line_len, stream))
	{
	    tokens = str_split(line, ',');
	    
	    for(int l = 0; l<c; l++){
			data[i*c + l] = atof(tokens[l]);
			
		}
		i++;
	}
	
	
	//-----------------------WEIGHT MATRIX---------------------
	
	
	float *W = (float *)malloc(numClasses * numClasses * c * sizeof(float));
	float *time = (float *)malloc(sizeof(float));
	time[0] = 0;
	//-----------------------SERIAL PEGASOS--------------------- 
	/*
	The following for loop extracts the relevant X and Y data.
	This data is passed into pegafit along with 
	the single weight vector for that particular
	comparison.
	Pegafit trains the weights for the particular 
	comparison.
	*/
	StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    
    
    
	float alpha = 0.0001;
	
	for(int j = 1; j<numClasses; j++){
		for(int k = 0; k<j; k++){
			
		
			int trainRows = classSizes[j*2 + 1] + classSizes[k*2 + 1];
			float *train_X = (float *)malloc(trainRows * c * sizeof(float));
			float *train_y = (float *)malloc(trainRows * sizeof(float));
			
			getModelData(data, train_X, train_y, classSizes, j, k);
			
			pegaFit(train_X, train_y, &W[j*numClasses*c+k*c+0], T, alpha, trainRows, time,timer);
			
		}
	}
	
	sdkStopTimer(&timer);
	float tot_time = sdkGetTimerValue(&timer);
    printf("Total time      :\t%f(ms)\n", tot_time);
    printf("Processing time :\t%f(ms)\n", time[0]);
    printf("Overheads time  :\t%f(ms)\n", tot_time-time[0]);
    sdkDeleteTimer(&timer);
	
	stream = fopen(filenameTrain, "r");
	i = 0;
	while (fgets(line, line_len, stream))
	{
	    tokens = str_split(line, ',');
	    
	    for(int l = 0; l<c; l++){
			data[i*c + l] = atof(tokens[l]);
			
		}
		i++;
	}
	
	
	int numCorr=numCorrect(numRecords, data, W,classSizes);
	
	float avg = (numCorr/(numRecords+0.0))*100;
	printf("Correct Percentage: %f\n",avg);
}
